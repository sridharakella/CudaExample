#include <studio.h>

int main()
{
    int nDevices();
    
    hipGetDeviceCount(&nDevices)
    {
        for(int i=0 ; i<nDevices ; i ++)
        {
            hipDeviceProp_t prop;
            cudaDeviceProperties(&prop, i);
            printf("Device Number %d\n"+i);
            printf("Device name %s\n"+prop.name);
            


        }

    }

}